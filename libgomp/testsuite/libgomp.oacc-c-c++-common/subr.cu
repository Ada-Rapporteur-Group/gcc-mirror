#include "hip/hip_runtime.h"

extern "C" __global__ void
delay (clock_t * d_o, clock_t delay)
{
  clock_t start, ticks;

  start = clock ();

  ticks = 0;

  while (ticks < delay)
    ticks = clock () - start;
}

extern "C" __global__ void
delay2 (unsigned long *d_o, clock_t delay, unsigned long tid)
{
  clock_t start, ticks;

  start = clock ();

  ticks = 0;

  while (ticks < delay)
    ticks = clock () - start;

  d_o[0] = tid;
}

extern "C" __global__ void
sum (clock_t * d_o, int N)
{
  int i;
  clock_t sum;
  __shared__ clock_t ticks[32];

  sum = 0;

  for (i = threadIdx.x; i < N; i += blockDim.x)
    sum += d_o[i];

  ticks[threadIdx.x] = sum;

  syncthreads ();

  for (i = 16; i >= 1; i >>= 1)
    {
      if (threadIdx.x < i)
	ticks[threadIdx.x] += ticks[threadIdx.x + i];

      syncthreads ();
    }

  d_o[0] = ticks[0];
}

extern "C" __global__ void
mult (int n, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  for (i = 0; i < n; i++)
    y[i] = x[i] * x[i];
}
